#include "hip/hip_runtime.h"
/**
 * @file matrix_add.cu
 * @brief CUDA implementations for matrix addition operations.
 *
 * This file contains device and host functions for performing matrix addition on the GPU.
 *
 * @author Steven Kight
 * @date 2025-10-09
 */

#include "matrix_core.h"

#include <iostream>


/**
 * @brief CUDA kernel for element-wise addition of two matrices.
 *
 * Each thread computes one element of the output matrix r.
 *
 * @param[in] a Pointer to the first input matrix (device memory).
 * @param[in] b Pointer to the second input matrix (device memory).
 * @param[out] r Pointer to the output matrix (device memory).
 * @param[in] rows Number of rows in the matrices.
 * @param[in] cols Number of columns in the matrices.
 */
__global__ void matrix_add_kernel(float *a, float *b, float *r, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols)
        r[row * cols + col] = a[row * cols + col] + b[row * cols + col];
}

/**
 * @brief Host function to perform matrix addition using CUDA.
 *
 * This function checks matrix dimensions, allocates device memory, copies data to the device,
 * launches the CUDA kernel for matrix addition, and copies the result back to the host.
 *
 * @param[in] A Pointer to the first input matrix (host memory).
 * @param[in] B Pointer to the second input matrix (host memory).
 * @param[out] R Pointer to the result matrix (host memory).
 *
 * @note All matrices must have the same dimensions. If not, the function prints an error and returns.
 */
extern "C" void matrix_add_cuda(const Matrix *A, const Matrix *B, Matrix *R) {
    if (A->rows != B->rows || A->cols != B->cols) {
        std::cerr << "Matrix dimensions do not match for addition" << std::endl;
        return;
    }

    float *d_a, *d_b, *d_r;
    hipMalloc((void **)&d_a, A->rows * A->cols * sizeof(float));
    hipMalloc((void **)&d_b, B->rows * B->cols * sizeof(float));
    hipMalloc((void **)&d_r, R->rows * R->cols * sizeof(float));

    hipMemcpy(d_a, A->data, A->rows * A->cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, B->data, B->rows * B->cols * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((A->cols + 15) / 16, (A->rows + 15) / 16);
    matrix_add_kernel<<<gridSize, blockSize>>>(d_a, d_b, d_r, A->rows, A->cols);

    hipMemcpy(R->data, d_r, R->rows * R->cols * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_a); hipFree(d_b); hipFree(d_r);
}
