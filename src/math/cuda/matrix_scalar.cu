#include "hip/hip_runtime.h"
/**
 * @file matrix_scalar.cu
 * @brief CUDA implementations for matrix scalar operations.
 *
 * This file contains device and host functions for performing matrix scalar operations on the GPU.
 * Functions are designed for use in high-performance scientific computing and research applications.
 *
 * @author Steven Kight
 * @date 2025-10-09
 */

#include "matrix_core.h"


/**
 * @brief Kernel: multiply every element by a scalar.
 *
 * @param[in] a Device pointer to input matrix data
 * @param[in] scalar Scalar multiplier
 * @param[out] r Device pointer to output matrix data
 * @param[in] rows Number of rows
 * @param[in] cols Number of columns
 */
__global__ void matrix_scalar_multiply_kernel(float *a, float scalar, float *r, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols)
        r[row * cols + col] = a[row * cols + col] * scalar;
}

/**
 * @brief Kernel: divide every element by a scalar.
 *
 * @param[in] a Device pointer to input matrix data
 * @param[in] scalar Scalar divisor
 * @param[out] r Device pointer to output matrix data
 * @param[in] rows Number of rows
 * @param[in] cols Number of columns
 */
__global__ void matrix_scalar_divide_kernel(float *a, float scalar, float *r, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols)
        r[row * cols + col] = a[row * cols + col] / scalar;
}

/**
 * @brief Kernel: add a scalar to every element.
 *
 * @param[in] a Device pointer to input matrix data
 * @param[in] scalar Scalar to add
 * @param[out] r Device pointer to output matrix data
 * @param[in] rows Number of rows
 * @param[in] cols Number of columns
 */
__global__ void matrix_scalar_add_kernel(float *a, float scalar, float *r, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols)
        r[row * cols + col] = a[row * cols + col] + scalar;
}

/**
 * @brief Kernel: subtract a scalar from every element.
 *
 * @param[in] a Device pointer to input matrix data
 * @param[in] scalar Scalar to subtract
 * @param[out] r Device pointer to output matrix data
 * @param[in] rows Number of rows
 * @param[in] cols Number of columns
 */
__global__ void matrix_scalar_subtract_kernel(float *a, float scalar, float *r, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols)
        r[row * cols + col] = a[row * cols + col] - scalar;
}

/**
 * @brief Host wrapper: multiply matrix by scalar using the GPU.
 *
 * This function allocates device memory, copies the input matrix to the device,
 * launches the CUDA kernel, and copies the result back to the host.
 *
 * @param[in] m Pointer to input matrix (host memory)
 * @param[in] scalar Scalar multiplier
 * @param[out] r Pointer to result matrix (host memory)
 */
extern "C" void matrix_scalar_multiply_cuda(const Matrix *m, float scalar, Matrix *r) {
    float *d_a, *d_r;
    hipMalloc((void **)&d_a, m->rows * m->cols * sizeof(float));
    hipMalloc((void **)&d_r, m->rows * m->cols * sizeof(float));

    hipMemcpy(d_a, m->data, m->rows * m->cols * sizeof(float), hipMemcpyHostToDevice);
    dim3 blockSize(16, 16);
    dim3 gridSize((m->cols + 15) / 16, (m->rows + 15) / 16);
    matrix_scalar_multiply_kernel<<<gridSize, blockSize>>>(d_a, scalar, d_r, m->rows, m->cols);
    hipMemcpy(r->data, d_r, m->rows * m->cols * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_a); hipFree(d_r);
}

/**
 * @brief Host wrapper: divide matrix by scalar using the GPU.
 *
 * This function allocates device memory, copies the input matrix to the device,
 * launches the CUDA kernel, and copies the result back to the host.
 *
 * @param[in] m Pointer to input matrix (host memory)
 * @param[in] scalar Scalar divisor
 * @param[out] r Pointer to result matrix (host memory)
 */
extern "C" void matrix_scalar_divide_cuda(const Matrix *m, float scalar, Matrix *r) {
    float *d_a, *d_r;
    hipMalloc((void **)&d_a, m->rows * m->cols * sizeof(float));
    hipMalloc((void **)&d_r, m->rows * m->cols * sizeof(float));

    hipMemcpy(d_a, m->data, m->rows * m->cols * sizeof(float), hipMemcpyHostToDevice);
    dim3 blockSize(16, 16);
    dim3 gridSize((m->cols + 15) / 16, (m->rows + 15) / 16);
    matrix_scalar_divide_kernel<<<gridSize, blockSize>>>(d_a, scalar, d_r, m->rows, m->cols);
    hipMemcpy(r->data, d_r, m->rows * m->cols * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_a); hipFree(d_r);
}

/**
 * @brief Host wrapper: add scalar to matrix using the GPU.
 *
 * This function allocates device memory, copies the input matrix to the device,
 * launches the CUDA kernel, and copies the result back to the host.
 *
 * @param[in] m Pointer to input matrix (host memory)
 * @param[in] scalar Scalar to add
 * @param[out] r Pointer to result matrix (host memory)
 */
extern "C" void matrix_scalar_add_cuda(const Matrix *m, float scalar, Matrix *r) {
    float *d_a, *d_r;
    hipMalloc((void **)&d_a, m->rows * m->cols * sizeof(float));
    hipMalloc((void **)&d_r, m->rows * m->cols * sizeof(float));

    hipMemcpy(d_a, m->data, m->rows * m->cols * sizeof(float), hipMemcpyHostToDevice);
    dim3 blockSize(16, 16);
    dim3 gridSize((m->cols + 15) / 16, (m->rows + 15) / 16);
    matrix_scalar_add_kernel<<<gridSize, blockSize>>>(d_a, scalar, d_r, m->rows, m->cols);
    hipMemcpy(r->data, d_r, m->rows * m->cols * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_a); hipFree(d_r);
}

/**
 * @brief Host wrapper: subtract scalar from matrix using the GPU.
 * This function allocates device memory, copies the input matrix to the device,
 * launches the CUDA kernel, and copies the result back to the host.
 *
 * @param[in] m Pointer to input matrix (host memory)
 * @param[in] scalar Scalar to subtract
 * @param[out] r Pointer to result matrix (host memory)
 */
extern "C" void matrix_scalar_subtract_cuda(const Matrix *m, float scalar, Matrix *r) {
    float *d_a, *d_r;
    hipMalloc((void **)&d_a, m->rows * m->cols * sizeof(float));
    hipMalloc((void **)&d_r, m->rows * m->cols * sizeof(float));

    hipMemcpy(d_a, m->data, m->rows * m->cols * sizeof(float), hipMemcpyHostToDevice);
    dim3 blockSize(16, 16);
    dim3 gridSize((m->cols + 15) / 16, (m->rows + 15) / 16);
    matrix_scalar_subtract_kernel<<<gridSize, blockSize>>>(d_a, scalar, d_r, m->rows, m->cols);
    hipMemcpy(r->data, d_r, m->rows * m->cols * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_a); hipFree(d_r);
}
