#include "hip/hip_runtime.h"
#include "matrix_core.h"

#include <iostream>


/**
 * @brief Kernel: element-wise subtraction R = A - B
 * 
 * This kernel computes the element-wise subtraction of two matrices A and B,
 * storing the result in matrix R. Each thread calculates one element of the
 * output matrix.
 *
 * @param[in] a Device pointer to first input matrix data
 * @param[in] b Device pointer to second input matrix data
 * @param[out] r Device pointer to output matrix data
 * @param[in] rows Number of rows
 * @param[in] cols Number of columns
 */
__global__ void matrix_subtract_kernel(float *a, float *b, float *r, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols)
        r[row * cols + col] = a[row * cols + col] - b[row * cols + col];
}

/**
 * @brief Host wrapper for matrix subtraction using the GPU.
 *
 * Validates dimensions, copies inputs to the device, launches the kernel,
 * and copies back the result.
 *
 * @param[in] A Pointer to first input matrix (host memory)
 * @param[in] B Pointer to second input matrix (host memory)
 * @param[out] R Pointer to result matrix (host memory)
 *
 * @note All matrices must have the same dimensions. If not, the function prints an error and returns.
 */
extern "C" void matrix_subtract_cuda(const Matrix *A, const Matrix *B, Matrix *R) {
    if (A->rows != B->rows || A->cols != B->cols) {
        std::cerr << "Matrix dimensions do not match for subtraction" << std::endl;
        return;
    }

    float *d_a, *d_b, *d_r;
    hipMalloc((void **)&d_a, A->rows * A->cols * sizeof(float));
    hipMalloc((void **)&d_b, B->rows * B->cols * sizeof(float));
    hipMalloc((void **)&d_r, R->rows * R->cols * sizeof(float));

    hipMemcpy(d_a, A->data, A->rows * A->cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, B->data, B->rows * B->cols * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((A->cols + 15) / 16, (A->rows + 15) / 16);
    matrix_subtract_kernel<<<gridSize, blockSize>>>(d_a, d_b, d_r, A->rows, A->cols);

    hipMemcpy(R->data, d_r, R->rows * R->cols * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_a); hipFree(d_b); hipFree(d_r);
}
