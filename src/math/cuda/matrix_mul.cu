#include "hip/hip_runtime.h"
/**
 * @file matrix_mul.cu
 * @brief CUDA implementations for matrix multiplication operations.
 *
 * This file contains device and host functions for performing matrix multiplication on the GPU.
 * Functions are designed for use in high-performance scientific computing and research applications.
 *
 * @author Steven Kight
 * @date 2025-10-09
 */
#include "matrix_core.h"

#include <iostream>


/**
 * @brief CUDA kernel performing matrix multiplication.
 *
 * Computes R = A * B using a straightforward inner product. Each thread
 * computes one element of the output matrix R.
 *
 * @param[in] a Device pointer to A data (rows_a x cols_a)
 * @param[in] rows_a Number of rows in A
 * @param[in] cols_a Number of cols in A
 * @param[in] b Device pointer to B data (rows_b x cols_b)
 * @param[in] rows_b Number of rows in B
 * @param[in] cols_b Number of cols in B
 * @param[out] r Device pointer to result data (rows_r x cols_r)
 * @param[in] rows_r Number of rows in result
 * @param[in] cols_r Number of cols in result
 */
__global__ void matrix_multiply_kernel(
    float *a, int rows_a, int cols_a, 
    float *b, int rows_b, int cols_b, 
    float *r, int rows_r, int cols_r
) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows_r && col < cols_r) {
        float sum = 0.0f;
        for (int k = 0; k < cols_a; k++) {
            sum += a[row * cols_a + k] * b[k * cols_b + col];
        }
        r[row * cols_r + col] = sum;
    }
}

/**
 * @brief Host wrapper that prepares device memory and launches the multiply kernel.
 *
 * This function validates input dimensions, allocates device buffers, copies
 * input data to device, launches the kernel, and retrieves the result.
 *
 * @param[in] A Pointer to left-hand matrix (host memory)
 * @param[in] B Pointer to right-hand matrix (host memory)
 * @param[out] R Pointer to result matrix (host memory)
 */
extern "C" void matrix_multiply_cuda(const Matrix *A, const Matrix *B, Matrix *R) {
    if (A->cols != B->rows) {
        std::cerr << "Matrix dimensions do not match for multiplication" << std::endl;
        return;
    }

    float *d_a, *d_b, *d_r;
    hipMalloc((void **)&d_a, A->rows * A->cols * sizeof(float));
    hipMalloc((void **)&d_b, B->rows * B->cols * sizeof(float));
    hipMalloc((void **)&d_r, R->rows * R->cols * sizeof(float));

    hipMemcpy(d_a, A->data, A->rows * A->cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, B->data, B->rows * B->cols * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((R->cols + 15) / 16, (R->rows + 15) / 16);
    matrix_multiply_kernel<<<gridSize, blockSize>>>(d_a, A->rows, A->cols, d_b, B->rows, B->cols, d_r, R->rows, R->cols);

    hipMemcpy(R->data, d_r, R->rows * R->cols * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_a); hipFree(d_b); hipFree(d_r);
}
