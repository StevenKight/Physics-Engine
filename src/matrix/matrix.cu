#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include "matrix.h"

__global__ void matrix_multiply_kernel(
    float *a, int rows_a, int cols_a, 
    float *b, int rows_b, int cols_b, 
    float *r, int rows_r, int cols_r
) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows_r && col < cols_r) {
        float sum = 0.0f;
        for (int k = 0; k < cols_a; k++) {
            sum += a[row * cols_a + k] * b[k * cols_b + col];
        }
        r[row * cols_r + col] = sum;
    }
}

__global__ void matrix_add_kernel(float *a, float *b, float *r, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        r[row * cols + col] = a[row * cols + col] + b[row * cols + col];
    }
}

__global__ void matrix_subtract_kernel(float *a, float *b, float *r, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        r[row * cols + col] = a[row * cols + col] - b[row * cols + col];
    }
}

extern "C" void matrix_multiply(const Matrix *A, const Matrix *B, Matrix *R) {
    // Allocate matrix dimensions
    int *rows_a = (int *)&A->rows;
    int *cols_a = (int *)&A->cols;
    int *rows_b = (int *)&B->rows;
    int *cols_b = (int *)&B->cols;
    int *rows_r = (int *)&R->rows;
    int *cols_r = (int *)&R->cols;
    
    // Check if dimensions match
    if (*cols_a != *rows_b) {
        std::cerr << "Matrix dimensions do not match for multiplication" << std::endl;
        return;
    }

    // Allocate device memory
    float *d_a, *d_b, *d_r;
    hipMalloc((void **)&d_a, A->rows * A->cols * sizeof(float));
    hipMalloc((void **)&d_b, B->rows * B->cols * sizeof(float));
    hipMalloc((void **)&d_r, R->rows * R->cols * sizeof(float));

    hipMemcpy(d_a, A->data, (*rows_a) * (*cols_a) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, B->data, (*rows_b) * (*cols_b) * sizeof(float), hipMemcpyHostToDevice);

    // Calculate grid and block sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((*cols_r + blockSize.x - 1) / blockSize.x, (*rows_r + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    matrix_multiply_kernel<<<gridSize, blockSize>>>(d_a, *rows_a, *cols_a, d_b, *rows_b, *cols_b, d_r, *rows_r, *cols_r);

    // Copy result back to host
    hipMemcpy(R->data, d_r, (*rows_r) * (*cols_r) * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_r);
}

extern "C" void matrix_add(const Matrix *A, const Matrix *B, Matrix *R) {
    // Allocate matrix dimensions
    int *rows_a = (int *)&A->rows;
    int *cols_a = (int *)&A->cols;
    int *rows_b = (int *)&B->rows;
    int *cols_b = (int *)&B->cols;
    int *rows_r = (int *)&R->rows;
    int *cols_r = (int *)&R->cols;

    // Check if dimensions match
    if (*rows_a != *rows_b || *cols_a != *cols_b) {
        std::cerr << "Matrix dimensions do not match for addition" << std::endl;
        return;
    }

    // Allocate device memory
    float *d_a, *d_b, *d_r;
    hipMalloc((void **)&d_a, A->rows * A->cols * sizeof(float));
    hipMalloc((void **)&d_b, B->rows * B->cols * sizeof(float));
    hipMalloc((void **)&d_r, R->rows * R->cols * sizeof(float));

    hipMemcpy(d_a, A->data, (*rows_a) * (*cols_a) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, B->data, (*rows_b) * (*cols_b) * sizeof(float), hipMemcpyHostToDevice);

    // Calculate grid and block sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((*cols_a + blockSize.x - 1) / blockSize.x, (*rows_a + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    matrix_add_kernel<<<gridSize, blockSize>>>(d_a, d_b, d_r, *rows_a, *cols_a);

    // Copy result back to host
    hipMemcpy(R->data, d_r, (*rows_r) * (*cols_r) * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_r);
}

extern "C" void matrix_subtract(const Matrix *A, const Matrix *B, Matrix *R) {
    // Allocate matrix dimensions
    int *rows_a = (int *)&A->rows;
    int *cols_a = (int *)&A->cols;
    int *rows_b = (int *)&B->rows;
    int *cols_b = (int *)&B->cols;
    int *rows_r = (int *)&R->rows;
    int *cols_r = (int *)&R->cols;

    // Check if dimensions match
    if (*rows_a != *rows_b || *cols_a != *cols_b) {
        std::cerr << "Matrix dimensions do not match for subtraction" << std::endl;
        return;
    }

    // Allocate device memory
    float *d_a, *d_b, *d_r;
    hipMalloc((void **)&d_a, A->rows * A->cols * sizeof(float));
    hipMalloc((void **)&d_b, B->rows * B->cols * sizeof(float));
    hipMalloc((void **)&d_r, R->rows * R->cols * sizeof(float));

    hipMemcpy(d_a, A->data, (*rows_a) * (*cols_a) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, B->data, (*rows_b) * (*cols_b) * sizeof(float), hipMemcpyHostToDevice);

    // Calculate grid and block sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((*cols_a + blockSize.x - 1) / blockSize.x, (*rows_a + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    matrix_subtract_kernel<<<gridSize, blockSize>>>(d_a, d_b, d_r, *rows_a, *cols_a);

    // Copy result back to host
    hipMemcpy(R->data, d_r, (*rows_r) * (*cols_r) * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_r);
}

// Initialize a matrix
extern "C" void initialize_matrix(Matrix* matrix, int rows, int cols) {
    matrix->rows = rows;
    matrix->cols = cols;
    matrix->data = (float*)malloc(rows * cols * sizeof(float));
}

// Free matrix memory
extern "C" void free_matrix(Matrix* matrix) {
    if (matrix->data) {
        free(matrix->data);
        matrix->data = NULL;
    }
}

// Fill a matrix with a specific value
extern "C" void fill_matrix(Matrix* matrix, float value) {
    for (int i = 0; i < matrix->rows * matrix->cols; i++) {
        matrix->data[i] = value;
    }
}

// Print a matrix to the console
extern "C" void print_matrix(const Matrix* matrix) {
    for (int i = 0; i < matrix->rows; i++) {
        for (int j = 0; j < matrix->cols; j++) {
            printf("%.2f ", matrix->data[i * matrix->cols + j]);
        }
        printf("\n");
    }
}